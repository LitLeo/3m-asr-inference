#include "cublas_common.h"

#include <hip/hip_runtime_api.h>
#include <algorithm>
#include <cassert>
#include <memory>
#include <numeric>
#include <stdexcept>
#include <vector>

#include "hipblas.h"
#include "hip/hip_fp16.h"
#include "hip/hip_runtime.h"
#include "common.h"

namespace nvinfer1 {
namespace plugin {

hipblasStatus_t cublasGemm(hipblasHandle_t handle, hipblasOperation_t transa, hipblasOperation_t transb,
                          int m, int n, int k,
                          const float alpha, const float* A, const float* B, 
                          const float beta, float* C) {
  int lda = k;
  if (transa == HIPBLAS_OP_T) lda = m;
  int ldb = n;
  if (transb == HIPBLAS_OP_T) ldb = k;
  int ldc = n;

  auto status = hipblasSgemm(handle, transb, transa, 
                            n, m, k, 
                            &alpha, B, ldb, A, lda,
                            &beta, C, ldc);
  return status;
}

hipblasStatus_t cublasGemm(hipblasHandle_t handle, hipblasOperation_t transa, hipblasOperation_t transb,
                          int m, int n, int k,
                          const float alpha, const float* A, int lda, const float* B, int ldb,
                          const float beta, float* C, int ldc) {
  auto status = hipblasSgemm(handle, transb, transa, 
                            n, m, k, 
                            &alpha, B, ldb, A, lda,
                            &beta, C, ldc);
  return status;
}

hipblasStatus_t cublasGemm(hipblasHandle_t handle, hipblasOperation_t transa, hipblasOperation_t transb,
                          int m, int n, int k,
                          const half alpha, const half* A, int lda, const half* B, int ldb,
                          const half beta, half* C, int ldc) {
  auto status = hipblasHgemm(handle, transb, transa, 
                            n, m, k, 
                            &alpha, B, ldb, A, lda,
                            &beta, C, ldc);
  return status;
}

hipblasStatus_t cublasGemm(hipblasHandle_t handle, hipblasOperation_t transa, hipblasOperation_t transb,
                          int m, int n, int k,
                          const float alpha, const char* A, int lda, const char* B, int ldb,
                          const float beta, int* C, int ldc) {
  auto status = hipblasGemmEx(handle, transb, transa, 
                             n, m, k, 
                             &alpha, B, HIP_R_8I, ldb, A, HIP_R_8I, lda,
                             &beta, C, HIP_R_32I, ldc, 
                             HIP_R_32I, HIPBLAS_GEMM_DEFAULT);
  return status;
}


}  // namespace plugin
}  // namespace nvinfer1


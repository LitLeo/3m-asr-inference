// Copyright (C) 2022 THL A29 Limited, a Tencent company. All rights reserved.
//
// Licensed under the BSD 3-Clause License (the "License"); you may not use this file except
// in compliance with the License. You may obtain a copy of the License at
//
// https://opensource.org/licenses/BSD-3-Clause
//
// Unless required by applicable law or agreed to in writing, software distributed
// under the License is distributed on an "AS IS" BASIS, WITHOUT WARRANTIES OR
// CONDITIONS OF ANY KIND, either express or implied. See the License for the
// specific language governing permissions and limitations under the License.

#include "cublas_common.h"

#include <hip/hip_runtime_api.h>
#include <algorithm>
#include <cassert>
#include <memory>
#include <numeric>
#include <stdexcept>
#include <vector>

#include "hipblas.h"
#include "hip/hip_fp16.h"
#include "hip/hip_runtime.h"
#include "common.h"

namespace nvinfer1 {
namespace plugin {

hipblasStatus_t cublasGemm(hipblasHandle_t handle, hipblasOperation_t transa, hipblasOperation_t transb,
                          int m, int n, int k,
                          const float alpha, const float* A, const float* B, 
                          const float beta, float* C) {
  int lda = k;
  if (transa == HIPBLAS_OP_T) lda = m;
  int ldb = n;
  if (transb == HIPBLAS_OP_T) ldb = k;
  int ldc = n;

  auto status = hipblasSgemm(handle, transb, transa, 
                            n, m, k, 
                            &alpha, B, ldb, A, lda,
                            &beta, C, ldc);
  return status;
}

hipblasStatus_t cublasGemm(hipblasHandle_t handle, hipblasOperation_t transa, hipblasOperation_t transb,
                          int m, int n, int k,
                          const float alpha, const float* A, int lda, const float* B, int ldb,
                          const float beta, float* C, int ldc) {
  auto status = hipblasSgemm(handle, transb, transa, 
                            n, m, k, 
                            &alpha, B, ldb, A, lda,
                            &beta, C, ldc);
  return status;
}

hipblasStatus_t cublasGemm(hipblasHandle_t handle, hipblasOperation_t transa, hipblasOperation_t transb,
                          int m, int n, int k,
                          const half alpha, const half* A, int lda, const half* B, int ldb,
                          const half beta, half* C, int ldc) {
  auto status = hipblasHgemm(handle, transb, transa, 
                            n, m, k, 
                            &alpha, B, ldb, A, lda,
                            &beta, C, ldc);
  return status;
}

hipblasStatus_t cublasGemm(hipblasHandle_t handle, hipblasOperation_t transa, hipblasOperation_t transb,
                          int m, int n, int k,
                          const float alpha, const char* A, int lda, const char* B, int ldb,
                          const float beta, int* C, int ldc) {
  auto status = hipblasGemmEx(handle, transb, transa, 
                             n, m, k, 
                             &alpha, B, HIP_R_8I, ldb, A, HIP_R_8I, lda,
                             &beta, C, HIP_R_32I, ldc, 
                             HIP_R_32I, HIPBLAS_GEMM_DEFAULT);
  return status;
}


}  // namespace plugin
}  // namespace nvinfer1

